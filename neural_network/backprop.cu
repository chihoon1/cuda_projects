#include <hip/hip_runtime.h>
#include <studio.h>
#include <stdlib.h>
#include <math.h>

// For feedforward NN
// list of tensors(2d) where each tensor represents a layer from the first to the last layer
// also, list includes activation layer but with weight zero.
// tensor serves as weight to a neuron in the prev layer to a neuron in the next layer
// and also as a adjacency matrix of a graph where non-zero weigth represents connection
//      row: outgoing neuron(prev).   col: incoming neuron(next)

// compute loss. Outer loop == epoch
//  In a middle loop where looping over layers
//      In one iteration, each weight is handled by one processor in a SM in GPU (P)
//      But if more processors than all num_weights in a layer,
//               use remaining processors to compute loss with the same weights but different data (P)
//  Once loss is computed, compute gradient with another middle loop (P)
//  use trainable weights as adjacency matrix to compute gradient (don't forget activation derivative)
//  Apply gradients parallel way by using one processor in a SM per one weight (P)

// Key: if more blocks needed than num_blocks limit, then split kernel function calls in multiple times
//          to use less than limited block amount per a kernel call while utilizing all processors

class 